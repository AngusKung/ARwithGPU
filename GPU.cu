#include "hip/hip_runtime.h"
#include <vector>
//#include <map>
#include "planeObject.h"
#include "supervoxel.h"
#include "planeVoxel.h"
#include "planesVector.h"
#include "SyncedMemory.h"
//cuda
#include <thrust/device_vector.h>
//#include <thrust/copy.h>
#define EXE_PER_THREAD 100
#define parrallel_threshold 0.8
#define mu 0.2
#define parrallel_filter 0.8
#define distance_to_plane 0.005

__device__ void push_back_voxel(planeObject& o, planeVoxel& v) {
    planeVoxel* ptr = o.begin;
    if(ptr == NULL) {
        ptr = &v;
        o.size++;
        return;
    }
    while(ptr->next != NULL) {
        ptr = ptr->next;
    }
    ptr->next = &v;
    o.size++;
    return;
}

__device__ void push_back_plane(planesVector& v, planeObject& o) {
    planeObject* ptr = v.begin;
    if(ptr == NULL) {
        ptr = &o;
        v.size++;
        return;
    }
    while(ptr->next != NULL) {
        ptr = ptr->next;
    }
    ptr->next = &o;
    v.size++;
    return;
}

__device__ void append(planeObject& be_appended, planeObject& to_append) {
    planeVoxel* end = be_appended.begin;
    while(end->next != NULL) end = end->next;
    end->next = to_append.begin;
    be_appended.size += to_append.size;
}

__device__ void
findNeighbor(planeObject& plane, supervoxel* v_ptr,
             double& the_normal_x, double& the_normal_y, double& the_normal_z,
             double* normal_vector_x, double* normal_vector_y, double* normal_vector_z)
{
  planeVoxel temp(v_ptr->cluster_num, v_ptr->cluster_int);
  push_back_voxel(plane, temp );

//==========ORIGINAL CODE===============
  for(int i = 0; i != v_ptr->n_size; ++i) {
    supervoxel* neighbor = v_ptr->neighbors[i];
    int neighbor_cluster_int = neighbor->cluster_int;
    // Check whether the neighbor has normals like a plane
    // Supervoxel has normal of 1
    if(the_normal_x * normal_vector_x[neighbor_cluster_int] + the_normal_y * normal_vector_y[neighbor_cluster_int] +
        the_normal_z * normal_vector_z[neighbor_cluster_int] > parrallel_threshold && neighbor->used == false){
      neighbor->used = true;
      the_normal_x = (1-mu)*the_normal_x+mu*normal_vector_x[neighbor_cluster_int];
      the_normal_y = (1-mu)*the_normal_y+mu*normal_vector_y[neighbor_cluster_int];
      the_normal_z = (1-mu)*the_normal_z+mu*normal_vector_z[neighbor_cluster_int];
      findNeighbor(plane, neighbor,the_normal_x,the_normal_y,the_normal_z,
                   normal_vector_x, normal_vector_y, normal_vector_z);
    }
  }
  return;
//====================================

/*==============fake code=============
  for(int i = 0; i != 7; ++i) {
    uint32_t neighbor_cluster = i;
    int neighbor_cluster_int = i;
    // Check whether the neighbor has normals like a plane
    // Supervoxel has normal of 1
    //double adj_parrallel_threshold = parrallel_threshold * (1+(pos_z[neighbor_cluster_int] - min_z));
    if(the_normal_x * normal_vector_x[neighbor_cluster_int] + the_normal_y * normal_vector_y[neighbor_cluster_int] +
        the_normal_z * normal_vector_z[neighbor_cluster_int] > parrallel_threshold ){
      the_normal_x = (1-mu)*the_normal_x+mu*normal_vector_x[neighbor_cluster_int];
      the_normal_y = (1-mu)*the_normal_y+mu*normal_vector_y[neighbor_cluster_int];
      the_normal_z = (1-mu)*the_normal_z+mu*normal_vector_z[neighbor_cluster_int];
      //findNeighbor(plane, ,the_normal_x,the_normal_y,the_normal_z,
        //           normal_vector_x, normal_vector_y, normal_vector_z);
                   //supervoxel_adjacency, clusters_int, clusters_used);
    }
  }
======================================*/
}

__global__ void labelWithGPU(supervoxel* voxels, int v_size,
                             double* normal_vector_x,
                             double* normal_vector_y,
                             double* normal_vector_z,
                             double* pos_x,
                             double* pos_y,
                             double* pos_z,
                             planesVector* p_v) {
  int base = threadIdx.x*EXE_PER_THREAD;
  int the_cluster_int;
  int size_temp;
  double the_normal_x, the_normal_y, the_normal_z, avn_x, avn_y, avn_z, avp_x, avp_y, avp_z; 
  planesVector planesVectors = p_v[threadIdx.x];
  for(int i = base; i != EXE_PER_THREAD+base; ++i)
  {
    if( i >= v_size ) break;
    if( voxels[i].used==true ) continue;
    voxels[i].used = true;
    the_cluster_int = voxels[i].cluster_int;
    the_normal_x = normal_vector_x[the_cluster_int];
    the_normal_y = normal_vector_y[the_cluster_int];
    the_normal_z = normal_vector_z[the_cluster_int];

    planeObject plane;
    findNeighbor(plane, &voxels[i], the_normal_x,the_normal_y,the_normal_z,
                 normal_vector_x, normal_vector_y, normal_vector_z);
    size_temp = plane.size;//planeObject pointer's
    if(size_temp <= 1)
      continue;
    avp_x = 0; avp_y = 0; avp_z = 0; avn_x = 0; avn_y = 0; avn_z = 0;
    planeVoxel* pv_it = plane.begin;
    while(pv_it != NULL) {
      the_cluster_int = pv_it->id;
      avn_x += normal_vector_x[the_cluster_int];
      avn_y += normal_vector_y[the_cluster_int];
      avn_z += normal_vector_z[the_cluster_int];
      avp_x += pos_x[the_cluster_int];
      avp_y += pos_y[the_cluster_int];
      avp_z += pos_z[the_cluster_int];
      pv_it = pv_it->next;
    }
    avn_x /= double(size_temp);
    avn_y /= double(size_temp);
    avn_z /= double(size_temp);
    avp_x /= double(size_temp);
    avp_y /= double(size_temp);
    avp_z /= double(size_temp);

        bool new_plane = true;
        //Planar Refinements
    for(int j = 0; j != blockDim.x; ++j) {
        planeObject* po_ptr = p_v[j].begin;
        while(po_ptr != NULL) {
            const double on_x = po_ptr->aver_nor_x;
            const double on_y = po_ptr->aver_nor_y;
            const double on_z = po_ptr->aver_nor_z;
            const double op_x = po_ptr->aver_pos_x;
            const double op_y = po_ptr->aver_pos_y;
            const double op_z = po_ptr->aver_pos_z;
            if(std::abs(avn_x*on_x +avn_y*on_y +avn_z*on_z) > parrallel_filter && 
               std::abs((avn_x*avp_x + avn_y*avp_y + avn_z*avp_z) - (on_x*op_x + on_y*op_y + on_z*op_z)) < distance_to_plane ) {
                new_plane = false;
                double weight = size_temp / double(size_temp + po_ptr->size);
                po_ptr->aver_nor_x = (1-weight)*on_x + weight*avn_x ;
                po_ptr->aver_nor_y = (1-weight)*on_y + weight*avn_y ;
                po_ptr->aver_nor_z = (1-weight)*on_z + weight*avn_z ;
                po_ptr->aver_pos_x = (1-weight)*op_x + weight*avp_x ;
                po_ptr->aver_pos_y = (1-weight)*op_y + weight*avp_y ;
                po_ptr->aver_pos_z = (1-weight)*op_z + weight*avp_z ;
                append(*po_ptr, plane);
                break;
            }
        po_ptr = po_ptr->next;
        }
        if(new_plane == false) break;
    }
        if(new_plane == true){
        plane.aver_nor_x = avn_x;
        plane.aver_nor_y = avn_y;
        plane.aver_nor_z = avn_z;
        plane.aver_pos_x = avp_x;
        plane.aver_pos_y = avp_y;
        plane.aver_pos_z = avp_z;
        
        push_back_plane( planesVectors, plane );
        }
  }
}

void copyPlaneToHost(planesVector* planesVectors_gpu, int numOfThreads, std::vector<planeObject>& planesVectors) {
    for(int i = 0; i != numOfThreads; ++i) {
        planeObject* ptr = planesVectors_gpu[i].begin;
        while(ptr->next != NULL) {
            planesVectors.push_back(*ptr);
            ptr = ptr->next;
        }
    }
}

__global__ void test_t(t) {
    int hi = 5;
    for(test* i = t->)
}

void gpu(const std::vector<supervoxel>& voxels,
         const std::vector<double>& normal_vector_x,
         const std::vector<double>& normal_vector_y,
         const std::vector<double>& normal_vector_z,
         const std::vector<double>& pos_x,
         const std::vector<double>& pos_y,
         const std::vector<double>& pos_z,
         std::vector<planeObject>& planesVectors
) {
/*//test code======
  MemoryBuffer<test> t(5);
  auto t_sync = t.CreateSync(5);
  test* t_gpu = t_sync.get_gpu_rw();
  test_t<<<1,1>>>(t_gpu);


//================
*/

  int v_size;//nvx_size, nvy_size, nvz_size, px_size, py_size, pz_size, pv_size;
  //read only
  thrust::device_vector<supervoxel> voxels_gpu_v(voxels.begin(), voxels.end());
  supervoxel* voxels_gpu = thrust::raw_pointer_cast(&voxels_gpu_v[0]);
  v_size = voxels_gpu_v.size();
/*
  size_t s_a_size = sizeOfMultiMap(supervoxel_adjacency);
  std::multimap<uint32_t, uint32_t>* supervoxel_adjacency_cpu = &supervoxel_adjacency;
  std::multimap<uint32_t, uint32_t>* supervoxel_adjacency_gpu;
  hipMalloc((void**) &supervoxel_adjacency_gpu, s_a_size);
  hipMemcpy(supervoxel_adjacency_gpu, &supervoxel_adjacency, s_a_size, hipMemcpyHostToDevice);
*/
  thrust::device_vector<double> normal_vector_x_gpu_v(normal_vector_x.begin(), normal_vector_x.end());
  double* normal_vector_x_gpu = thrust::raw_pointer_cast(&normal_vector_x_gpu_v[0]);
  thrust::device_vector<double> normal_vector_y_gpu_v(normal_vector_y.begin(), normal_vector_y.end());
  double* normal_vector_y_gpu = thrust::raw_pointer_cast(&normal_vector_y_gpu_v[0]);
  thrust::device_vector<double> normal_vector_z_gpu_v(normal_vector_z.begin(), normal_vector_z.end());
  double* normal_vector_z_gpu = thrust::raw_pointer_cast(&normal_vector_z_gpu_v[0]);

  thrust::device_vector<double> pos_x_gpu_v(pos_x.begin(), pos_x.end());
  double* pos_x_gpu = thrust::raw_pointer_cast(&pos_x_gpu_v[0]);
  thrust::device_vector<double> pos_y_gpu_v(pos_y.begin(), pos_y.end());
  double* pos_y_gpu = thrust::raw_pointer_cast(&pos_y_gpu_v[0]);
  thrust::device_vector<double> pos_z_gpu_v(pos_z.begin(), pos_z.end());
  double* pos_z_gpu = thrust::raw_pointer_cast(&pos_z_gpu_v[0]);
  //read and write
  //thrust::device_vector<planeObject> planesVectors_gpu_v;
  //planeObject* planesVectors_gpu = thrust::raw_pointer_cast(&planesVectors_gpu_v[0]);
//  pv_size = planesVectors_gpu_v.size();
  //start labeling
  int numOfThreads = ( v_size % EXE_PER_THREAD)? v_size/EXE_PER_THREAD+1: v_size/EXE_PER_THREAD;

  MemoryBuffer<planesVector> pv(numOfThreads);
  auto pv_sync = pv.CreateSync(numOfThreads);
  planesVector* planesVectors_cpu = pv_sync.get_cpu_rw();
  planesVector* planesVectors_gpu = pv_sync.get_gpu_rw();
 // hipMalloc((void**) &planesVectors_gpu, numOfThreads);
 // hipMemcpy(planesVectors_gpu, planesVectors_cpu, numOfThreads, hipMemcpyHostToDevice);
  labelWithGPU<<<1, numOfThreads>>>(voxels_gpu, v_size,
                                    normal_vector_x_gpu,  normal_vector_y_gpu, normal_vector_z_gpu,
                                    pos_x_gpu, pos_y_gpu, pos_z_gpu, planesVectors_gpu);
 // hipMemcpy(planesVectors_cpu, planesVectors_gpu, numOfThreads, hipMemcpyDeviceToHost);
  copyPlaneToHost(planesVectors_cpu, numOfThreads, planesVectors);
  //thrust::copy(planesVectors_gpu_v.begin(), planesVectors_gpu_v.end(), planesVectors.begin());
}
