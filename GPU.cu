#include "hip/hip_runtime.h"
#include <vector>
//#include <map>
#include "planeObject.h"
#include "supervoxel.h"
//#include "planeVoxel.h"
//#include "planesVector.h"
#include "SyncedMemory.h"
//cuda
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
//#include <thrust/generate.h>
//#include <thrust/sort.h>
//#include <thrust/copy.h>
#define EXE_PER_THREAD 100
#define parrallel_threshold 0.8
#define mu 0.2
#define parrallel_filter 0.8
#define distance_to_plane 0.005
/*
__device__ void push_back_voxel(planeObject& o, planeVoxel& v) {
    planeVoxel* ptr = o.begin;
    if(ptr == NULL) {
        ptr = &v;
        o.size++;
        return;
    }
    while(ptr->next != NULL) {
        ptr = ptr->next;
    }
    ptr->next = &v;
    o.size++;
    return;
}

__device__ void push_back_plane(planesVector& v, planeObject& o) {
    planeObject* ptr = v.begin;
    if(ptr == NULL) {
        ptr = &o;
        v.size++;
        return;
    }
    while(ptr->next != NULL) {
        ptr = ptr->next;
    }
    ptr->next = &o;
    v.size++;
    return;
}

__device__ void append(planeObject& be_appended, planeObject& to_append) {
    planeVoxel* end = be_appended.begin;
    while(end->next != NULL) end = end->next;
    end->next = to_append.begin;
    be_appended.size += to_append.size;
}
*/
__device__ void resetPlane(planeObject* plane) {
                plane->size = 0;
                plane->aver_nor_x = 0;
                plane->aver_nor_y = 0;
                plane->aver_nor_z = 0;
                plane->aver_pos_x = 0;
                plane->aver_pos_y = 0;
                plane->aver_pos_z = 0;
}
__device__ void
//todo: add plane_id
findNeighbor(planeObject& plane, supervoxel* v_ptr, const int& plane_id,
             double& the_normal_x, double& the_normal_y, double& the_normal_z,
             double* normal_vector_x, double* normal_vector_y, double* normal_vector_z,
             double* pos_x, double* pos_y, double* pos_z)
{
//todo: plane.size++, v_ptr->plane_id = plane_id
  plane.size++;
  v_ptr->plane_id = plane_id;
//todo: cluster_int = v_ptr->cluster_int;
  int cluster_int = v_ptr->cluster_int;
//todo: plane.aver_nor_x += normal_vector_x[cluster_int], ..., plane.aver_pos_x += pos_x[cluster_int]
  plane.aver_nor_x += normal_vector_x[cluster_int];
  plane.aver_nor_y += normal_vector_y[cluster_int];
  plane.aver_nor_z += normal_vector_z[cluster_int];
  plane.aver_pos_x += pos_x[cluster_int];
  plane.aver_pos_y += pos_y[cluster_int];
  plane.aver_pos_z += pos_z[cluster_int];

  //planeVoxel temp(v_ptr->cluster_num, v_ptr->cluster_int);
  //push_back_voxel(plane, temp );

//==========ORIGINAL CODE===============
  for(int i = 0; i != v_ptr->n_size; ++i) {
    supervoxel* neighbor = v_ptr->neighbors[i];
    int neighbor_cluster_int = neighbor->cluster_int;
    // Check whether the neighbor has normals like a plane
    // Supervoxel has normal of 1
    if(the_normal_x * normal_vector_x[neighbor_cluster_int] + the_normal_y * normal_vector_y[neighbor_cluster_int] +
        the_normal_z * normal_vector_z[neighbor_cluster_int] > parrallel_threshold && neighbor->used == false){
      neighbor->used = true;
      the_normal_x = (1-mu)*the_normal_x+mu*normal_vector_x[neighbor_cluster_int];
      the_normal_y = (1-mu)*the_normal_y+mu*normal_vector_y[neighbor_cluster_int];
      the_normal_z = (1-mu)*the_normal_z+mu*normal_vector_z[neighbor_cluster_int];
      findNeighbor(plane, neighbor, plane_id, the_normal_x,the_normal_y,the_normal_z,
                   normal_vector_x, normal_vector_y, normal_vector_z,
                   pos_x, pos_y, pos_z);
    }
  }
  return;
//====================================

/*==============fake code=============
  for(int i = 0; i != 7; ++i) {
    uint32_t neighbor_cluster = i;
    int neighbor_cluster_int = i;
    // Check whether the neighbor has normals like a plane
    // Supervoxel has normal of 1
    //double adj_parrallel_threshold = parrallel_threshold * (1+(pos_z[neighbor_cluster_int] - min_z));
    if(the_normal_x * normal_vector_x[neighbor_cluster_int] + the_normal_y * normal_vector_y[neighbor_cluster_int] +
        the_normal_z * normal_vector_z[neighbor_cluster_int] > parrallel_threshold ){
      the_normal_x = (1-mu)*the_normal_x+mu*normal_vector_x[neighbor_cluster_int];
      the_normal_y = (1-mu)*the_normal_y+mu*normal_vector_y[neighbor_cluster_int];
      the_normal_z = (1-mu)*the_normal_z+mu*normal_vector_z[neighbor_cluster_int];
      //findNeighbor(plane, ,the_normal_x,the_normal_y,the_normal_z,
        //           normal_vector_x, normal_vector_y, normal_vector_z);
                   //supervoxel_adjacency, clusters_int, clusters_used);
    }
  }
======================================*/
}

__global__ void labelWithGPU(supervoxel* voxels, int v_size,
                             double* normal_vector_x,
                             double* normal_vector_y,
                             double* normal_vector_z,
                             double* pos_x,
                             double* pos_y,
                             double* pos_z,
                             planeObject* p_v) {
  int base = threadIdx.x*EXE_PER_THREAD; //for voxels
  int plane_id = threadIdx.x*EXE_PER_THREAD/2; //for p_v
  int pv_size = blockDim.x*EXE_PER_THREAD/2;
  int the_cluster_int;
  int size_temp;
  double the_normal_x, the_normal_y, the_normal_z, avn_x, avn_y, avn_z, avp_x, avp_y, avp_z; 
  for(int i = base; i != EXE_PER_THREAD+base; ++i)
  {
    if( i >= v_size ) break;
    if( voxels[i].used==true ) continue;
    voxels[i].used = true;
    the_cluster_int = voxels[i].cluster_int;
    the_normal_x = normal_vector_x[the_cluster_int];
    the_normal_y = normal_vector_y[the_cluster_int];
    the_normal_z = normal_vector_z[the_cluster_int];

    planeObject* plane = &p_v[plane_id];
//todo: add plane_id
    findNeighbor(*plane, &voxels[i], plane_id, the_normal_x,the_normal_y,the_normal_z,
                 normal_vector_x, normal_vector_y, normal_vector_z,
                 pos_x, pos_y, pos_z);
    size_temp = plane->size;
    if(size_temp <= 1) {
      for(int j = 0; j != v_size; ++j) {
        if(voxels[j].plane_id == plane_id) voxels[j].plane_id = -1;
        break;
      }
      resetPlane(plane);
      continue;
    }
/*//delete=======
    avp_x = 0; avp_y = 0; avp_z = 0; avn_x = 0; avn_y = 0; avn_z = 0;
    planeVoxel* pv_it = plane.begin;
    while(pv_it != NULL) {
      the_cluster_int = pv_it->id;
      avn_x += normal_vector_x[the_cluster_int];
      avn_y += normal_vector_y[the_cluster_int];
      avn_z += normal_vector_z[the_cluster_int];
      avp_x += pos_x[the_cluster_int];
      avp_y += pos_y[the_cluster_int];
      avp_z += pos_z[the_cluster_int];
      pv_it = pv_it->next;
    }
*///============
//modify to avn_x = plane->aver_nor_x/(double)size_temp, ...
    avn_x = plane->aver_nor_x/double(size_temp);
    avn_y = plane->aver_nor_y/double(size_temp);
    avn_z = plane->aver_nor_z/double(size_temp);
    avp_x = plane->aver_pos_x/double(size_temp);
    avp_y = plane->aver_pos_y/double(size_temp);
    avp_z = plane->aver_pos_z/double(size_temp);

        bool new_plane = true;
        //Planar Refinements
//for and while modify to one for through the whole planesVectors
    for(int j = 0; j != pv_size; ++j) {
        if(p_v[j].size == 0) continue;
        const double on_x = p_v[j].aver_nor_x;
        const double on_y = p_v[j].aver_nor_y;
        const double on_z = p_v[j].aver_nor_z;
        const double op_x = p_v[j].aver_pos_x;
        const double op_y = p_v[j].aver_pos_y;
        const double op_z = p_v[j].aver_pos_z;
        if(std::abs(avn_x*on_x +avn_y*on_y +avn_z*on_z) > parrallel_filter && 
           std::abs((avn_x*avp_x + avn_y*avp_y + avn_z*avp_z) - (on_x*op_x + on_y*op_y + on_z*op_z)) < distance_to_plane ) {
           new_plane = false;
           double weight = size_temp / double(size_temp + p_v[j].size);
           p_v[j].aver_nor_x = (1-weight)*on_x + weight*avn_x ;
           p_v[j].aver_nor_y = (1-weight)*on_y + weight*avn_y ;
           p_v[j].aver_nor_z = (1-weight)*on_z + weight*avn_z ;
           p_v[j].aver_pos_x = (1-weight)*op_x + weight*avp_x ;
           p_v[j].aver_pos_y = (1-weight)*op_y + weight*avp_y ;
           p_v[j].aver_pos_z = (1-weight)*op_z + weight*avp_z ;
//todo: update the size of the plane to be appended, and reset the variables of plane
           p_v[j].size += plane->size;
           resetPlane(plane);
           //append(*po_ptr, plane);
           break;
        }
    }
    if(new_plane == true){
//todo: plane_id++
        plane->aver_nor_x = avn_x;
        plane->aver_nor_y = avn_y;
        plane->aver_nor_z = avn_z;
        plane->aver_pos_x = avp_x;
        plane->aver_pos_y = avp_y;
        plane->aver_pos_z = avp_z;
        
        plane_id++;
        //push_back_plane( planesVectors, plane );
    }
  }
}

void copyPlaneToHost(planeObject* planesVectors_cpu, const int& pv_size, std::vector<planeObject>& planesVectors) {
//todo: replace for and while with a for through the whole planesVectors_cpu
    for(int i = 0; i != pv_size; ++i) {
        if(planesVectors_cpu[i].size == 0) continue;
        planesVectors.push_back(planesVectors_cpu[i]);
    }
    return;
}
/*
__global__ void test_t(t) {
    int hi = 5;
    for(test* i = t->)
}
*/
void gpu(const std::vector<supervoxel>& voxels,
         const std::vector<double>& normal_vector_x,
         const std::vector<double>& normal_vector_y,
         const std::vector<double>& normal_vector_z,
         const std::vector<double>& pos_x,
         const std::vector<double>& pos_y,
         const std::vector<double>& pos_z,
         std::vector<planeObject>& planesVectors
) {
/*//test code======
  MemoryBuffer<test> t(5);
  auto t_sync = t.CreateSync(5);
  test* t_gpu = t_sync.get_gpu_rw();
  test_t<<<1,1>>>(t_gpu);


//================
*/

  int v_size;//nvx_size, nvy_size, nvz_size, px_size, py_size, pz_size, pv_size;
  //read only
  thrust::device_vector<supervoxel> voxels_gpu_v(voxels.begin(), voxels.end());
  supervoxel* voxels_gpu = thrust::raw_pointer_cast(&voxels_gpu_v[0]);
  v_size = voxels_gpu_v.size();
/*
  size_t s_a_size = sizeOfMultiMap(supervoxel_adjacency);
  std::multimap<uint32_t, uint32_t>* supervoxel_adjacency_cpu = &supervoxel_adjacency;
  std::multimap<uint32_t, uint32_t>* supervoxel_adjacency_gpu;
  hipMalloc((void**) &supervoxel_adjacency_gpu, s_a_size);
  hipMemcpy(supervoxel_adjacency_gpu, &supervoxel_adjacency, s_a_size, hipMemcpyHostToDevice);
*/
  thrust::device_vector<double> normal_vector_x_gpu_v(normal_vector_x.begin(), normal_vector_x.end());
  double* normal_vector_x_gpu = thrust::raw_pointer_cast(&normal_vector_x_gpu_v[0]);
  thrust::device_vector<double> normal_vector_y_gpu_v(normal_vector_y.begin(), normal_vector_y.end());
  double* normal_vector_y_gpu = thrust::raw_pointer_cast(&normal_vector_y_gpu_v[0]);
  thrust::device_vector<double> normal_vector_z_gpu_v(normal_vector_z.begin(), normal_vector_z.end());
  double* normal_vector_z_gpu = thrust::raw_pointer_cast(&normal_vector_z_gpu_v[0]);

  thrust::device_vector<double> pos_x_gpu_v(pos_x.begin(), pos_x.end());
  double* pos_x_gpu = thrust::raw_pointer_cast(&pos_x_gpu_v[0]);
  thrust::device_vector<double> pos_y_gpu_v(pos_y.begin(), pos_y.end());
  double* pos_y_gpu = thrust::raw_pointer_cast(&pos_y_gpu_v[0]);
  thrust::device_vector<double> pos_z_gpu_v(pos_z.begin(), pos_z.end());
  double* pos_z_gpu = thrust::raw_pointer_cast(&pos_z_gpu_v[0]);
  //read and write
  //thrust::device_vector<planeObject> planesVectors_gpu_v;
  //planeObject* planesVectors_gpu = thrust::raw_pointer_cast(&planesVectors_gpu_v[0]);
//  pv_size = planesVectors_gpu_v.size();
  //start labeling
  int numOfThreads = ( v_size % EXE_PER_THREAD)? v_size/EXE_PER_THREAD+1: v_size/EXE_PER_THREAD;

  int pv_size = numOfThreads*EXE_PER_THREAD/2;
  MemoryBuffer<planeObject> pv(pv_size);
  auto pv_sync = pv.CreateSync(pv_size);
  planeObject* planesVectors_cpu = pv_sync.get_cpu_rw();
  planeObject* planesVectors_gpu = pv_sync.get_gpu_rw();
 // hipMalloc((void**) &planesVectors_gpu, numOfThreads);
 // hipMemcpy(planesVectors_gpu, planesVectors_cpu, numOfThreads, hipMemcpyHostToDevice);
  labelWithGPU<<<1, numOfThreads>>>(voxels_gpu, v_size,
                                    normal_vector_x_gpu,  normal_vector_y_gpu, normal_vector_z_gpu,
                                    pos_x_gpu, pos_y_gpu, pos_z_gpu, planesVectors_gpu);
 // hipMemcpy(planesVectors_cpu, planesVectors_gpu, numOfThreads, hipMemcpyDeviceToHost);
  copyPlaneToHost(planesVectors_cpu, pv_size, planesVectors);
  //thrust::copy(planesVectors_gpu_v.begin(), planesVectors_gpu_v.end(), planesVectors.begin());
  return;
}
